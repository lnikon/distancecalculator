
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <time.h>

void random_ints(int* a, const int n);
void print(int* a, const int n);

#define TOTAL 32
#define BLOCKS 8
#define THREADS 4

#define BLOCK_SIZE 8
#define RADIUS 3

__global__ void add(int* a, int* b, int* c, const int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        c[index] = a[index] + b[index];
    }
}

__global__ void stencil_1d(int* in, int* out)
{
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int            gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int            lindex = threadIdx.x + RADIUS;

    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS)
    {
        temp[lindex - RADIUS]     = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; ++offset)
    {
        result += temp[lindex + offset];
    }

    out[gindex] = result;
}

int main(void)
{
    srand(time(NULL));

    int *     a, *b, *c;
    int *     d_a, *d_b, *d_c;
    const int size = TOTAL * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    random_ints(a, TOTAL);
    b = (int*)malloc(size);
    random_ints(b, TOTAL);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    const int threadCount = THREADS;
    const int blockCount  = TOTAL / threadCount;

    std::cout << "\nBlockCount: " << blockCount << "\nThreadCount: " << threadCount << "\n";

    add<<<blockCount, threadCount>>>(d_a, d_b, d_c, TOTAL);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    print(a, TOTAL);
    print(b, TOTAL);
    print(c, TOTAL);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}

void random_ints(int* a, const int n)
{
    for (int i = 0; i < n; ++i)
        a[i] = rand() % 100;
}

void print(int* a, const int n)
{
    for (int i = 0; i < n; ++i)
        std::cout << std::setw(4) << a[i] << ' ';
    std::cout << std::endl;
}
