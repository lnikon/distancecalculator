#include "hip/hip_runtime.h"
#pragma CUDA_MATRIX_DIFFERENCE

#include <stdio.h>
#include <assert.h>

#include "MatrixDifference.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void L1Distance(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount) {
	int row = threadIdx.x + blockDim.x * blockIdx.x;

	if (row < queryRowCount) {
		for (int dRow = 0; dRow < datasetRowCount; dRow++) {
			for (int column = 0; column < columnCount; ++column) {
				int dcc = dRow * columnCount + column;
				int qcc = row * columnCount + column;
				int drcc = row * queryRowCount * columnCount + dcc;
				distances[drcc] = dataset[dcc] - query[qcc];
			}
		}
	}
}

__global__ void L2Distance(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount) {
	int row = threadIdx.x + blockDim.x * blockIdx.x;

	if (row < queryRowCount) {
		for (int dRow = 0; dRow < datasetRowCount; dRow++) {
			for (int column = 0; column < columnCount; ++column) {
				int dcc = dRow * columnCount + column;
				int qcc = row * columnCount + column;
				int drcc = row * queryRowCount * columnCount + dcc;
				int imm = dataset[dcc] - query[qcc];
				distances[drcc] = imm * imm;
			}
		}
	}
}

__global__ void HammingDistance(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount) {
	int row = threadIdx.x + blockDim.x * blockIdx.x;

	if (row < queryRowCount) {
		for (int dRow = 0; dRow < datasetRowCount; dRow++) {
			for (int column = 0; column < columnCount; ++column) {
				int dcc = dRow * columnCount + column;
				int qcc = row * columnCount + column;
				int drcc = row * queryRowCount * columnCount + dcc;
				int imm = dataset[dcc] - query[qcc];
				distances[drcc] = imm * imm;
			}
		}
	}
}

void L1DistanceRunner(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount)
{
	const int rowCountDistances = queryRowCount * datasetRowCount;
	const int querySize = queryRowCount * columnCount;
	const int datasetSize = datasetRowCount * columnCount;
	const int distancesSize = rowCountDistances * columnCount;

	float* deviceQuery;
	gpuErrchk(hipMalloc(&deviceQuery, sizeof(float) * querySize));
	gpuErrchk(hipMemcpy(deviceQuery, query, sizeof(float) * querySize, hipMemcpyHostToDevice));

	float* deviceDataset;
	gpuErrchk(hipMalloc(&deviceDataset, sizeof(float) * datasetSize));
	gpuErrchk(hipMemcpy(deviceDataset, dataset, sizeof(float) * datasetSize, hipMemcpyHostToDevice));

	float* deviceDistances;
	gpuErrchk(hipMalloc(&deviceDistances, sizeof(float) * distancesSize));

	const int blockSize = 1024;
	const int gridRows = (rowCountDistances + blockSize - 1) / blockSize;
	dim3 gridDim(gridRows);
	dim3 blockDim(blockSize);

	L1Distance<<<gridDim, blockDim>>>(deviceQuery, deviceDataset, deviceDistances, queryRowCount, datasetRowCount, columnCount);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(distances, deviceDistances, sizeof(float) * distancesSize, hipMemcpyDeviceToHost));

	hipFree(deviceQuery);
	hipFree(deviceDataset);
	hipFree(deviceDistances);
}

void L2DistanceRunner(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount)
{
	const int rowCountDistances = queryRowCount * datasetRowCount;
	const int querySize = queryRowCount * columnCount;
	const int datasetSize = datasetRowCount * columnCount;
	const int distancesSize = rowCountDistances * columnCount;

	float* deviceQuery;
	gpuErrchk(hipMalloc(&deviceQuery, sizeof(float) * querySize));
	gpuErrchk(hipMemcpy(deviceQuery, query, sizeof(float) * querySize, hipMemcpyHostToDevice));

	float* deviceDataset;
	gpuErrchk(hipMalloc(&deviceDataset, sizeof(float) * datasetSize));
	gpuErrchk(hipMemcpy(deviceDataset, dataset, sizeof(float) * datasetSize, hipMemcpyHostToDevice));

	float* deviceDistances;
	gpuErrchk(hipMalloc(&deviceDistances, sizeof(float) * distancesSize));

	const int blockSize = 1024;
	const int gridRows = (rowCountDistances + blockSize - 1) / blockSize;
	dim3 gridDim(gridRows);
	dim3 blockDim(blockSize);

	L2Distance<<<gridDim, blockDim>>>(deviceQuery, deviceDataset, deviceDistances, queryRowCount, datasetRowCount, columnCount);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(distances, deviceDistances, sizeof(float) * distancesSize, hipMemcpyDeviceToHost));

	hipFree(deviceQuery);
	hipFree(deviceDataset);
	hipFree(deviceDistances);
}

void HammingDistanceRunner(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount)
{
	const int rowCountDistances = queryRowCount * datasetRowCount;
	const int querySize = queryRowCount * columnCount;
	const int datasetSize = datasetRowCount * columnCount;
	const int distancesSize = rowCountDistances * columnCount;

	float* deviceQuery;
	gpuErrchk(hipMalloc(&deviceQuery, sizeof(float) * querySize));
	gpuErrchk(hipMemcpy(deviceQuery, query, sizeof(float) * querySize, hipMemcpyHostToDevice));

	float* deviceDataset;
	gpuErrchk(hipMalloc(&deviceDataset, sizeof(float) * datasetSize));
	gpuErrchk(hipMemcpy(deviceDataset, dataset, sizeof(float) * datasetSize, hipMemcpyHostToDevice));

	float* deviceDistances;
	gpuErrchk(hipMalloc(&deviceDistances, sizeof(float) * distancesSize));

	const int blockSize = 1024;
	const int gridRows = (rowCountDistances + blockSize - 1) / blockSize;
	dim3 gridDim(gridRows);
	dim3 blockDim(blockSize);

	HammingDistance<<<gridDim, blockDim>>>(deviceQuery, deviceDataset, deviceDistances, queryRowCount, datasetRowCount, columnCount);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(distances, deviceDistances, sizeof(float) * distancesSize, hipMemcpyDeviceToHost));

	hipFree(deviceQuery);
	hipFree(deviceDataset);
	hipFree(deviceDistances);
}
