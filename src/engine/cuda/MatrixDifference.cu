#include "hip/hip_runtime.h"
#pragma CUDA_MATRIX_DIFFERENCE

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#include "MatrixDifference.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void L1Distance(int* query, int* dataset, int* distances, int queryRowCount, int datasetRowCount, int columnCount) {
	int row = threadIdx.x + blockDim.x * blockIdx.x;
	
	int queryIndex = row % queryRowCount;
	assert(queryIndex < queryRowCount);
	int datasetIndex = row % datasetRowCount;
	assert(datasetIndex < datasetRowCount);

	if (row < queryRowCount * datasetRowCount * columnCount) {
		for (int diffIdx = 0; diffIdx < columnCount; diffIdx++) {
			distances[row] = dataset[datasetIndex + diffIdx] - query[queryIndex + diffIdx];
		}
	}
}

void L1DistanceRunner()
{
	const int queryRowCount = 16;
	const int datasetRowCount = 2*queryRowCount;
	const int rowCountDistances = queryRowCount * datasetRowCount;
	const int columnCount = 16;

	const int querySize = queryRowCount * columnCount;
	int* query = new int[querySize];
	for (int i = 0; i < querySize; i++) {
		query[i] = 10;
	}

	const int datasetSize = datasetRowCount * columnCount;
	int* dataset = new int[datasetSize];
	for (int i = 0; i < datasetSize; i++) {
		dataset[i] = 20;
	}

	const int distancesSize = rowCountDistances * columnCount;
	int* distances = new int[distancesSize];
	int* correctDistances = new int[distancesSize];

	for (int i = 0; i < rowCountDistances; i++) {
		for (int j = 0; j < queryRowCount; j++) {
			for (int k = 0; k < datasetRowCount; k++) {
				for (int p = 0; p < columnCount; p++) {
					correctDistances[i + p] = dataset[k * datasetRowCount + p] - query[j * queryRowCount + p];
				}
			}
		}
	}

	int* deviceQuery;
	gpuErrchk(hipMalloc(&deviceQuery, sizeof(int) * querySize));
	gpuErrchk(hipMemcpy(deviceQuery, query, sizeof(int) * querySize, hipMemcpyHostToDevice));

	int* deviceDataset;
	gpuErrchk(hipMalloc(&deviceDataset, sizeof(int) * datasetSize));
	gpuErrchk(hipMemcpy(deviceDataset, dataset, sizeof(int) * datasetSize, hipMemcpyHostToDevice));

	int* deviceDistances;
	gpuErrchk(hipMalloc(&deviceDistances, sizeof(int) * distancesSize));

	const int blockSize = 1024;
	const int gridRows = (distancesSize + blockSize - 1) / blockSize;
	dim3 gridDim(gridRows);
	dim3 blockDim(blockSize);
	printf("gridDim: (%d)\n", gridRows);
	printf("blockDim: (%d)\n", blockSize);

	L1Distance<<<gridDim, blockDim>>>(deviceQuery, deviceDataset, deviceDistances, queryRowCount, datasetRowCount, columnCount);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(distances, deviceDistances, sizeof(int) * distancesSize, hipMemcpyDeviceToHost));

	hipFree(deviceQuery);
	hipFree(deviceDataset);
	hipFree(deviceDistances);
}
