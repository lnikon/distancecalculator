#include "hip/hip_runtime.h"
#pragma CUDA_MATRIX_DIFFERENCE

#include <stdio.h>
#include <assert.h>

#include "MatrixDifference.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void L1Distance(float* query, float* dataset, float* distances, int queryRowCount, int datasetRowCount, int columnCount) {
	int row = threadIdx.x + blockDim.x * blockIdx.x;
	
	int queryIndex = row % queryRowCount;
	assert(queryIndex < queryRowCount);
	int datasetIndex = row % datasetRowCount;
	assert(datasetIndex < datasetRowCount);

	if (row < queryRowCount * datasetRowCount * columnCount) {
		for (int diffIdx = 0; diffIdx < columnCount; diffIdx++) {
			distances[row] = dataset[datasetIndex + diffIdx] - query[queryIndex + diffIdx];
		}
	}
}

void L1DistanceRunner()
{
	const int queryRowCount = 16;
	const int datasetRowCount = 2*queryRowCount;
	const int rowCountDistances = queryRowCount * datasetRowCount;
	const int columnCount = 16;

	const int querySize = queryRowCount * columnCount;
	float* query = new float[querySize];
	for (int i = 0; i < querySize; i++) {
		query[i] = 10;
	}

	const int datasetSize = datasetRowCount * columnCount;
	float* dataset = new float[datasetSize];
	for (int i = 0; i < datasetSize; i++) {
		dataset[i] = 20;
	}

	const int distancesSize = rowCountDistances * columnCount;
	float* distances = new float[distancesSize];
	float* correctDistances = new float[distancesSize];

	for (int i = 0; i < rowCountDistances; i++) {
		for (int j = 0; j < queryRowCount; j++) {
			for (int k = 0; k < datasetRowCount; k++) {
				for (int p = 0; p < columnCount; p++) {
					correctDistances[i + p] = dataset[k * datasetRowCount + p] - query[j * queryRowCount + p];
				}
			}
		}
	}

	float* deviceQuery;
	gpuErrchk(hipMalloc(&deviceQuery, sizeof(float) * querySize));
	gpuErrchk(hipMemcpy(deviceQuery, query, sizeof(float) * querySize, hipMemcpyHostToDevice));

	float* deviceDataset;
	gpuErrchk(hipMalloc(&deviceDataset, sizeof(float) * datasetSize));
	gpuErrchk(hipMemcpy(deviceDataset, dataset, sizeof(float) * datasetSize, hipMemcpyHostToDevice));

	float* deviceDistances;
	gpuErrchk(hipMalloc(&deviceDistances, sizeof(float) * distancesSize));

	const int blockSize = 1024;
	const int gridRows = (distancesSize + blockSize - 1) / blockSize;
	dim3 gridDim(gridRows);
	dim3 blockDim(blockSize);
	printf("gridDim: (%d)\n", gridRows);
	printf("blockDim: (%d)\n", blockSize);

	L1Distance<<<gridDim, blockDim>>>(deviceQuery, deviceDataset, deviceDistances, queryRowCount, datasetRowCount, columnCount);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(distances, deviceDistances, sizeof(float) * distancesSize, hipMemcpyDeviceToHost));

	hipFree(deviceQuery);
	hipFree(deviceDataset);
	hipFree(deviceDistances);
}
